#include "hip/hip_runtime.h"
/*
* Simpleconvolution.cu
*
*   procedure CUDA CONVOLUTION(signal, kernel, K, L, M, norm)
*     cuMemcpy(gpu s, signal, HostToDevice)
*     cuMemcpy(gpu k, kernel, HostToDevice)
*     gpu s ← cuFFT(gpu s)
*     gpu k ← cuFFT(gpu k)
*     gpu s ← pwProd(gpu s, gpu k, K, L, M, norm)
*     gpu s ← cuIFFT(gpu s)
*     cuMemcpy(signal, gpu s, DeviceToHost)
*   end procedure
*
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


#define BATCH 10

int main()
{

  int *k_h, *s_d, *k_d;

  hipfftComplex *s_h;

  int size;

  hipfftComplex *data;

  // xyz dimension size of the sum of signals.
  int K, L, M;

  printf("Enter K, L, M\n");
  scanf("%d %d %d", &K, &L, &M);

  size = (sizeof(int) * (K + L + M));

  // Allocate host memory.
  /*s_h = (int *) malloc(sizeof(int) * size);
  k_h = (int *) malloc(sizeof(int) * size);

  // Temp.
  int k, l, m;

  // Generate both signal and kernel for now.
  for (k = 0; k < K; k++)
    for (l = 0; l < L; l++)
      for (m = 0; m < M; m++) {
        // First fill x, then y and then z.
        s_h[k + l + m] = m;
        k_h[k + l + m] = m;
      }*/

  s_h = (hipfftComplex *) malloc(sizeof(hipfftComplex) * K * BATCH);

  // Allocate Device Memory.
  hipMalloc((void **) &s_d, size);
  hipMalloc((void **) &data, (sizeof(hipfftComplex) * K * BATCH)); // ComplexData
  hipMalloc((void **) &k_d, size);

  // Copy to device.
  hipMemcpy(s_d, s_h, size, hipMemcpyHostToDevice);
  hipMemcpy(k_d, k_h, size, hipMemcpyHostToDevice);

  hipfftHandle plan = 1;

  // Do an Dimension first. DFT size of K.

  for (int i = 0; i < (sizeof(hipfftComplex) * K * BATCH); i++) {
    s_h[i].x = i;
    s_h[i].y = i;
  }

  hipMemcpy(data, s_h, (sizeof(hipfftComplex) * K * BATCH), hipMemcpyHostToDevice);

  hipfftPlan1d(&plan, K, HIPFFT_C2C, BATCH);

  hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);

  hipDeviceSynchronize();

  hipfftDestroy(plan);

  hipMemcpy(s_h, data, (sizeof(hipfftComplex) * K * BATCH), hipMemcpyDeviceToHost);

  hipFree(data);

  for (int i = 0; i < (sizeof(hipfftComplex) * K * BATCH); i++)
    printf("%f %f \n", s_h[i].x, s_h[i].y);

  // Copy results, back, from the GPU to the CPU.
  hipMemcpy(s_h, s_d, size, hipMemcpyDeviceToHost);

  // Cleanup
  free(s_h);
  free(k_h);
  hipFree(s_d);
  hipFree(k_d);

}

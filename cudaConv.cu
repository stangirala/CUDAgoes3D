#include "hip/hip_runtime.h"
#include <cudaConv.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


int checkCUDA() {

  if ((system("nvidia-settings -q gpus")) == 0) {

    int deviceCount;
    hipError_t e = hipGetDeviceCount(&deviceCount);
    if (e != hipSuccess) {
      return 1;
    }
  }
  else
    return 1;

  return 0;
}


void
printData(Complex *a, int size, char *msg) {

  if (msg == "") printf("\n");
  else printf("%s\n", msg);

  for (int i = 0; i < size; i++)
    printf("%f %f\n", a[i].x, a[i].y);
}

void
normData(Complex *a, int size, float norm) {

  for (int i = 0; i < size; i++) {
    a[i].x /= norm;
    a[i].y /= norm;
  }
}

// flag = 1 for real signals.
void
randomFill(Complex *h_signal, int size, int flag) {

  // Real signal.
  if (flag == REAL) {
    for (int i = 0; i < size; i++) {
      h_signal[i].x = rand() / (float) RAND_MAX;
      h_signal[i].y = 0;
    }
  }
}

// FFT a signal that's on the _DEVICE_.
void
signalFFT(Complex *d_signal, int signal_size) {

  // Handle type used to store and execute CUFFT plans.
  // Essentially allocates the resouecwes and sort of interns
  // them.

  hipfftHandle plan;
  if (hipfftPlan1d(&plan, signal_size, HIPFFT_C2C, 1) != HIPFFT_SUCCESS) {
    printf("Failed to plan FFT\n");
    exit(0);
  }

  // Execute the plan.
  if (hipfftExecC2C(plan, (hipfftComplex *) d_signal, (hipfftComplex *) d_signal, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
    printf ("Failed Executing FFT\n");
    exit(0);
  }

}


// Reverse of the signalFFT(.) function.
void
signalIFFT(Complex *d_signal, int signal_size) {

  hipfftHandle plan;
  if (hipfftPlan1d(&plan, signal_size, HIPFFT_C2C, 1) != HIPFFT_SUCCESS) {
    printf("Failed to plan IFFT\n");
    exit(0);
  }

  if (hipfftExecC2C(plan, (hipfftComplex *) d_signal, (hipfftComplex *) d_signal, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
    printf ("Failed Executing FFT\n");
    exit(0);
  }
}


// Pointwise Multiplication Kernel.
__global__ void
pwProd(Complex *signal1, int size1, Complex *signal2, int size2) {

  int threadsPerBlock, blockId, globalIdx;

  threadsPerBlock = blockDim.x * blockDim.y;
  blockId = blockIdx.x + (blockIdx.y * gridDim.x);
  globalIdx = (blockId * threadsPerBlock) + threadIdx.x + (threadIdx.y * blockDim.x);

  if (globalIdx <= size1) {

      signal1[globalIdx].x = (signal1[globalIdx].x * signal2[globalIdx].x - signal1[globalIdx].y * signal2[globalIdx].y);
      signal1[globalIdx].y = (signal1[globalIdx].x * signal2[globalIdx].y + signal1[globalIdx].y * signal2[globalIdx].x);
    }

}

void
cudaConvolution(Complex *d_signal1, int size1, Complex *d_signal2,
                int size2, dim3 blockSize, dim3 gridSize) {

  signalFFT(d_signal1, size1);
  signalFFT(d_signal2, size2);

  pwProd<<<gridSize, blockSize>>>(d_signal1, size1, d_signal2, size2);

  //signalIFFT(d_signal1, size1);

}


int allocateAndPad(Complex **a, int s1, Complex **b, int s2) {

  int oldsize, newsize, i;

  newsize = s1 + s2 - 1;

  while (!((newsize != 0) && !(newsize & (newsize - 1)))) {
    newsize++;
  }

  oldsize = s1;
  *a = (Complex *) malloc(sizeof(Complex) * newsize);
  for (i = oldsize; i < newsize; i++) {
    (*a)[i].x = 0;
    (*a)[i].y = 0;
  }

  oldsize = s2;
  *b = (Complex *) malloc(sizeof(Complex) * newsize);
  for (i = oldsize; i < newsize; i++) {
    (*b)[i].x = 0;
    (*b)[i].y = 0;
  }

  return newsize;
}

int main()
{

  Complex *h1, *h2, *d1, *d2;

  int s1, s2, newsize, i, dim;


  if (checkCUDA()) {
    printf ("CUDA FAIL\n");
    exit(0);
  }


  dim = 1;

  s1 = 16;
  s2 = 16;

  for (i = 0; i < dim; i++)  {

      newsize = allocateAndPad(&h1, s1, &h2, s2);

      randomFill(h1, s1, REAL);
      randomFill(h2, s2, REAL);

      // Kernel Block and Grid Size.
      const dim3 blockSize(16, 16, 1);
      const dim3 gridSize(newsize / 16 + 1, newsize / 16 + 1, 1);

      printData(h1, newsize, "H Signal 1");
      printData(h2, newsize, "H Signal 2");

      hipMalloc(&d1, sizeof(Complex) * newsize);
      hipMalloc(&d2, sizeof(Complex) * newsize);
      hipMemcpy(d1, h1, sizeof(Complex) * newsize, hipMemcpyHostToDevice);
      hipMemcpy(d2, h2, sizeof(Complex) * newsize, hipMemcpyHostToDevice);

      cudaConvolution(d1, newsize, d2, newsize, blockSize, gridSize);

      //hipDeviceSynchronize();

      hipMemcpy(h1, d1, sizeof(Complex) * newsize, hipMemcpyDeviceToHost);

      normData(h1, newsize, newsize);

      printData(h1, newsize, "Conv");

      free(h1); free(h2);
      hipFree(d1); hipFree(d2);

      hipDeviceReset();
  }

  return 0;
}

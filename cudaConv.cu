/*
* Simpleconvolution.cu
*
* procedure CUDA CONVOLUTION(signal, kernel, K, L, M, norm)
* cuMemcpy(gpu s, signal, HostToDevice)
* cuMemcpy(gpu k, kernel, HostToDevice)
* gpu s ← cuFFT(gpu s)
* gpu k ← cuFFT(gpu k)
* gpu s ← pwProd(gpu s, gpu k, K, L, M, norm)
* gpu s ← cuIFFT(gpu s)
* cuMemcpy(signal, gpu s, DeviceToHost)
* end procedure
*
*/


#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

typedef enum signaltype {REAL, COMPLEX} signal;

typedef float2 Complex;

void
printData(Complex *a, int size, char *msg) {

  if (msg == "") printf("\n");
  else printf("%s\n", msg);

  for (int i = 0; i < size; i++)
    printf("%f %f\n", a[i].x, a[i].y);
}

void
normData(Complex *a, int size, float norm) {

  for (int i = 0; i < size; i++) {
    a[i].x /= norm;
    a[i].y /= norm;
  }
}

// flag = 1 for real signals.
void
randomFill(Complex *h_signal, int size, int flag) {

  // Real signal.
  if (flag == REAL) {
    for (int i = 0; i < size; i++) {
      h_signal[i].x = rand() / (float) RAND_MAX;
      h_signal[i].y = 0;
    }
  }
}

// FFT a signal that's on the _DEVICE_.
void
signalFFT(Complex *d_signal, int signal_size) {

  // Handle type used to store and execute CUFFT plans.
  // Essentially allocates the resouecwes and sort of interns
  // them.

  hipfftHandle plan;
  if (hipfftPlan1d(&plan, signal_size, HIPFFT_C2C, 1) != HIPFFT_SUCCESS) {
    printf("Failed to plan FFT\n");
    exit(0);
  }

  // Execute the plan.
  if (hipfftExecC2C(plan, (hipfftComplex *) d_signal, (hipfftComplex *) d_signal, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
    printf ("Failed Executing FFT\n");
    exit(0);
  }

}


// Reverse of the signalFFT(.) function.
void
signalIFFT(Complex *d_signal, int signal_size) {

  hipfftHandle plan;
  if (hipfftPlan1d(&plan, signal_size, HIPFFT_C2C, 1) != HIPFFT_SUCCESS) {
    printf("Failed to plan IFFT\n");
    exit(0);
  }

  if (hipfftExecC2C(plan, (hipfftComplex *) d_signal, (hipfftComplex *) d_signal, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
    printf ("Failed Executing FFT\n");
    exit(0);
  }
}


// Pointwise Multiplication Kernel.
__global__ void
pwProd(Complex *signal1, int size1, Complex *signal2, int size2) {

  int threadsPerBlock, blockId, globalIdx;

  threadsPerBlock = blockDim.x * blockDim.y;
  blockId = blockIdx.x + (blockIdx.y * gridDim.x);
  globalIdx = (blockId * threadsPerBlock) + threadIdx.x + (threadIdx.y * blockDim.x);

  if (globalIdx < size1) {
      signal1[globalIdx].x = signal1[globalIdx].x * signal2[globalIdx].x - signal1[globalIdx].y * signal2[globalIdx].y;
      signal1[globalIdx].y = signal1[globalIdx].x * signal2[globalIdx].y + signal1[globalIdx].y * signal2[globalIdx].x;
    }

}

void
cudaConvolution(Complex *d_signal1, int size1, Complex *d_signal2,
                int size2, dim3 blockSize, dim3 gridSize) {

  signalFFT(d_signal1, size1);
  signalFFT(d_signal2, size2);

  pwProd<<<gridSize, blockSize>>>(d_signal1, size1, d_signal2, size2);

  signalIFFT(d_signal1, size1);

}


// factor represents how the DIC algorithm is applied on the convolution.
// That is, a factor of 16 implies a 16 way split on the signal and a 16 way
// call on the convolution.
// Assuming both signals are of the same size.
void
cudaConvolutionDIC(Complex *d_signal1, int size1, Complex *d_signal2, int size2, dim3 blockSize, dim3 gridSize, int load) {

  // TODO Padding!

  int i;

  if (load >= size1) {
    cudaConvolution(d_signal1, load, d_signal2, load, blockSize, gridSize);
  }
  else{
    for (i = 0; i < size1; i++)
      cudaConvolution((d_signal1 + i * load), load, (d_signal2 + i * load), load, blockSize, gridSize);
  }
}

int main()
{

  Complex *h_signal, *d_signal1, *d_signal2;

  int alloc_size;

  alloc_size = 16;

  // Kernel Block and Grid Size.
  const dim3 blockSize(16, 16, 1);
  const dim3 gridSize(alloc_size / 16 + 1, alloc_size / 16 + 1, 1);

  h_signal = (Complex *) malloc(sizeof(Complex) * alloc_size);

  hipMalloc(&d_signal1, sizeof(Complex) * alloc_size);
  if (hipGetLastError() != hipSuccess){
    printf("Cuda error: Failed to allocate\n");
    exit(0);
  }
  hipMalloc(&d_signal2, sizeof(Complex) * alloc_size);

  // Add random data to signal.
  randomFill(h_signal, alloc_size, REAL);

  printData(h_signal, alloc_size, "Random H1");
  hipMemcpy(d_signal1, h_signal, sizeof(Complex) * alloc_size, hipMemcpyHostToDevice);

  printData(h_signal, alloc_size, "Random H2");
  hipMemcpy(d_signal2, h_signal, sizeof(Complex) * alloc_size, hipMemcpyHostToDevice);

  cudaConvolution(d_signal1, alloc_size, d_signal2, alloc_size, blockSize, gridSize);

  hipDeviceSynchronize();

  hipMemcpy(h_signal, d_signal1, sizeof(Complex) * alloc_size, hipMemcpyDeviceToHost);
  normData(h_signal, alloc_size, alloc_size);
  printData(h_signal, alloc_size, "IFFT");

  return 0;
}
